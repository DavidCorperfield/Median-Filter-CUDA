#include "hip/hip_runtime.h"
#include "../include/filter.hpp"

using namespace std;

/* ============================================DEVICE INFO============================================
  CUDA Driver Version / Runtime Version          6.5 / 6.5
  CUDA Capability Major/Minor version number:    3.0
  Total amount of global memory:                 4096 MBytes (4294770688 bytes)
  ( 8) Multiprocessors, (192) CUDA Cores/MP:     1536 CUDA Cores
  GPU Clock rate:                                797 MHz (0.80 GHz)
  Memory Clock rate:                             2500 Mhz
  Memory Bus Width:                              256-bit
  L2 Cache Size:                                 524288 bytes
  Maximum Texture Dimension Size (x,y,z)         1D=(65536), 2D=(65536, 65536), 3D=(4096, 4096, 4096)
  Maximum Layered 1D Texture Size, (num) layers  1D=(16384), 2048 layers
  Maximum Layered 2D Texture Size, (num) layers  2D=(16384, 16384), 2048 layers
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       49152 bytes
  Total number of registers available per block: 65536
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  2048
  Maximum number of threads per block:           1024
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
  Maximum memory pitch:                          2147483647 bytes
  Texture alignment:                             512 bytes
  Concurrent copy and kernel execution:          Yes with 2 copy engine(s)
  Run time limit on kernels:                     No
  Integrated GPU sharing Host Memory:            No
  Support host page-locked memory mapping:       Yes
  Alignment requirement for Surfaces:            Yes
  Device has ECC support:                        Disabled
  Device supports Unified Addressing (UVA):      Yes
  Device PCI Bus ID / PCI location ID:           0 / 3
  Compute Mode: < Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >

  deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 6.5, CUDA Runtime Version = 6.5, NumDevs = 1, Device0 = GRID K520
*/

/**
 * Gets the global thread index of a 2D Grid of 2D blocks.
 */
__device__
inline uint get_global_thread_index(const uint width) {
    const uint x_index      = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y_index      = blockIdx.y * blockDim.y + threadIdx.y;
    const uint pixel_one_d  = x_index + y_index * width;
    return pixel_one_d;
}

__global__
void kernel_median_filter(const uint filter_size, const uchar * device_input_data, uchar * device_output_data, const uint height, const uint width) {
    const uint offset        = (filter_size - 1) / 2;
    const uint filter_length = filter_size * filter_size;
    const uint thread_index  = get_global_thread_index(width);
    const uint x             = thread_index / width;
    const uint y             = thread_index % width;

    // Allocate memory for the filter array
    uchar * filter_array     = new uchar[filter_length];

    // Init the filter array with 0 or 255 values
    // Will write over the indices that are VIEWABLE from the context pixel
#pragma unroll
    for (uint i = 0; i < filter_length; ++i) {
        filter_array[i] = i % 2 == 0 ? MIN_RGB_VALUE : MAX_RGB_VALUE;
    }

    const uchar * context  = device_input_data  + thread_index;
    uchar * output_context = device_output_data + thread_index;

    // Populate the filter_array
    uint filter_array_index = 0;

#pragma unroll
    for (int y_offset = -1 * static_cast<int>(offset); y_offset <= static_cast<int>(offset); ++y_offset) {
#pragma unroll
        for (int x_offset = -1 * static_cast<int>(offset); x_offset <= static_cast<int>(offset); ++x_offset) {
            // Handle special cases for when the offset would place us beyond the bounds of the input.
            const int x_focus = x + x_offset;
            const int y_focus = y + y_offset;

            // Check if one of the neighboring pixels of our context pixel is outside the grid
            if (x_focus < 0 || x_focus >= width || y_focus < 0 || y_focus >= height) {
                continue;
            }
            // Otherwise we're not an edge or corner, so we have all of our neighbors
            filter_array[filter_array_index++] = *(context + static_cast<int>(x_offset) + static_cast<int>(width) * static_cast<int>(y_offset));
        }
    }

    // Sort the filter_array.
    // TODO: If had CUDA 7.0, we'd be using Thrust on the device.
    // But, we don't right now, so just do a Selection Sort.
    uchar swap;
    uint min_index;
#pragma unroll
    for (uint i = 0; i < filter_length - 1; ++i) {
        min_index = i;
#pragma unroll
        for (uint j = i + 1; j < filter_length; ++j) {
            if (filter_array[j] < filter_array[min_index])
                min_index = j;
        }
        swap = filter_array[min_index];
        filter_array[min_index] = filter_array[i];
        filter_array[i] = swap;
    }

    // Grab the median. Note that the since we always had odd window sizes,
    // then filter_size * filter_size is always odd as well - so no need to
    // handle special cases for even or odd number for the median.
    *output_context = filter_array[(filter_length - 1) / 2];
}

double Filter::median_filter_gpu(const uint filter_size, const uchar * host_data, uchar * output, const uint height, const uint width) {
    const int size = height * width * sizeof(uchar);

    /* Allocate device memory for the result. */
    /* Note that output to hold the HOST memory has already been allocated for. */
    void * device_input_data  = nullptr;
    void * device_output_data = nullptr;
    checkCudaErrors(hipMalloc((void **) & device_input_data, size));
    checkCudaErrors(hipMalloc((void **) & device_output_data, size));

    /* Copy the input data to the device. */
    checkCudaErrors(hipMemcpy(
            device_input_data,      // dst
            host_data,              // src
            size,                   // count
            hipMemcpyHostToDevice
    ));

    /* Launch the kernel! */
    dim3 grid(GRID_X, GRID_Y, 1);
    dim3 block(BLOCK_X, BLOCK_Y, 1);
    kernel_median_filter<<< grid, block >>>(filter_size, (uchar *) device_input_data, (uchar *) device_output_data, height, width);

    /* In case the kernel had problems, I'd like to know. */
    checkCudaErrors(hipGetLastError());

    /* At this point, we just need to copy the device output data back to the host memory. */
    checkCudaErrors(hipMemcpy(
            output,                 // dst
            device_output_data,     // src
            size,                   // count
            hipMemcpyDeviceToHost
    ));

    hipFree(device_input_data);
    hipFree(device_output_data);

    /* Capture the device copy-compute-copy time. */
    return get_timer_value();
}

void Filter::median_filter_cpu(const uint filter_size, const uchar * input, uchar * output, const uint height, const uint width) {
    // How far in any which direction you can go.
    // (-1, -1) (0, -1) (1, -1)
    // (-1,  0) (0,  0) (1,  0)
    // (-1,  1) (0,  1) (1,  1)
    const uint offset = (filter_size - 1) / 2;
    const uint filter_length = filter_size * filter_size;

    // For testing purposes, print out RGB values of original Lena image.
    #ifdef LENA
        for (uint y = 0; y < height; ++y) {
            for (uint x = 0; x < width; ++x) {
                cout << static_cast<int>(input[x + width * y]) << " ";
            }
            cout << endl;
        }
    #endif

    /**
     * Iterate and perform median filter analysis on every pixel.
     * Make outer loop the ys so that successive reads are as close to each other as possible,
     * i.e. for single-threaded CPU code, it is most important for caching, but for GPUs it is
     * most important for coalesced memory access (and maybe caching).
     * If we iterate over the rows first, we have 0 coalescing then.
     */
    for (uint y = 0; y < height; ++y) {
        for (uint x = 0; x < width; ++x) {

            uchar filter_array[filter_length];
            // Init the filter array with 0 or 255 values
            // Will write over the indices that are VIEWABLE from the context pixel
            for (uint i = 0; i < filter_length; ++i) {
                filter_array[i] = i % 2 == 0 ? MIN_RGB_VALUE : MAX_RGB_VALUE;
            }

            // What pixel am I currently looking at
            const uchar * context  = &input[x + width * y];
            uchar * output_context = &output[x + width * y];

    	    // Populate the filter_array.
            uint filter_array_index = 0;

#pragma unroll
            for (int y_offset = -1 * static_cast<int>(offset); y_offset <= static_cast<int>(offset); ++y_offset) {
#pragma unroll
                for (int x_offset = -1 * static_cast<int>(offset); x_offset <= static_cast<int>(offset); ++x_offset) {
        		    // Handle special cases for when the offset would place us beyond the bounds of the input.
                    const int x_focus = x + x_offset;
                    const int y_focus = y + y_offset;

                    // Check if one of the neighboring pixels of our context pixel is outside the grid
                    if (x_focus < 0 || x_focus >= width || y_focus < 0 || y_focus >= height) {
                        continue;
                    }
                    // Otherwise we're not an edge or corner, so we have all of our neighbors
                    filter_array[filter_array_index++] = *(context + static_cast<int>(x_offset) + static_cast<int>(width) * static_cast<int>(y_offset));
        		}
    	    }

    	    // Sort the filter_array.
            sort(filter_array, filter_array + filter_length);

            // Print the filter array to test.
            // #ifdef _DEBUG
            //     for (uint i = 0; i < filter_length - 1; ++i) {
            //         cout << static_cast<int>(filter_array[i]) << " ";
            //     }
            //     cout << static_cast<int>(filter_array[filter_length - 1]) << endl;
            // #endif

            // Grab the median. Note that the since we always had odd window sizes,
            // then filter_size * filter_size is always odd as well - so no need to
            // handle special cases for even or odd number for the median.
            *output_context = filter_array[(filter_length - 1) / 2];
	   }
    }
}

double Filter::median_filter_verify_errors(const uint filter_size, const uchar * input_data, const uchar * compare, const uint height, const uint width) {
    uchar * cpu_results = (uchar * ) malloc(height * width * sizeof(uchar));
    if (!cpu_results) {
        throw runtime_error("Problems in reserving memory for the CPU version.");
    }

    /* Do the Median Filter using the CPU. */
    median_filter_cpu(filter_size, input_data, cpu_results, height, width);

    const char * cpu_saved_file = "cpu_output.pgm";

    if (!sdkSavePGM<uchar>(cpu_saved_file, cpu_results, width, height)) {
        throw runtime_error("Error in saving the output image!");
    }
    cout << "Using the CPU version, we saved the image with filename: " << cpu_saved_file << endl;

    /* Walk through and compare the pixels of the images to see how many are wrong. */
    uint error_pixel_count = 0;
    for (uint i = 0; i < height * width; ++i) {
        if (cpu_results[i] != compare[i])
            ++error_pixel_count;
    }

    /* Return the percentage of how many pixels are wrong. */
    return error_pixel_count / (height * width);
}
