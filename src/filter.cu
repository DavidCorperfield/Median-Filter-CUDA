#include "../include/filter.hpp"

using namespace std;

/* ============================================DEVICE INFO============================================
  CUDA Driver Version / Runtime Version          6.5 / 6.5
  CUDA Capability Major/Minor version number:    3.0
  Total amount of global memory:                 4096 MBytes (4294770688 bytes)
  ( 8) Multiprocessors, (192) CUDA Cores/MP:     1536 CUDA Cores
  GPU Clock rate:                                797 MHz (0.80 GHz)
  Memory Clock rate:                             2500 Mhz
  Memory Bus Width:                              256-bit
  L2 Cache Size:                                 524288 bytes
  Maximum Texture Dimension Size (x,y,z)         1D=(65536), 2D=(65536, 65536), 3D=(4096, 4096, 4096)
  Maximum Layered 1D Texture Size, (num) layers  1D=(16384), 2048 layers
  Maximum Layered 2D Texture Size, (num) layers  2D=(16384, 16384), 2048 layers
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       49152 bytes
  Total number of registers available per block: 65536
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  2048
  Maximum number of threads per block:           1024
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
  Maximum memory pitch:                          2147483647 bytes
  Texture alignment:                             512 bytes
  Concurrent copy and kernel execution:          Yes with 2 copy engine(s)
  Run time limit on kernels:                     No
  Integrated GPU sharing Host Memory:            No
  Support host page-locked memory mapping:       Yes
  Alignment requirement for Surfaces:            Yes
  Device has ECC support:                        Disabled
  Device supports Unified Addressing (UVA):      Yes
  Device PCI Bus ID / PCI location ID:           0 / 3
  Compute Mode: < Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >

  deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 6.5, CUDA Runtime Version = 6.5, NumDevs = 1, Device0 = GRID K520
*/

double Filter::median_filter_gpu(const uint filter_size, const uchar * host_data, uchar * output, const uint height, const uint width) {
    const uint size = height * width * sizeof(uchar);

    /* Allocate device memory for the result. */
    uchar * device_data = nullptr;
    checkCudaErrors(hipMalloc((void **) & device_data, size));
    checkCudaErrors(
        hipMemcpy(
            device_data,            // dst
            host_data,              // src
            size,                   // count
            hipMemcpyHostToDevice
        )
    );

    return 0;
}

void Filter::median_filter_cpu(const uint filter_size, const uchar * input, uchar * output, const uint height, const uint width) {
    // How far in any which direction you can go.
    // (-1, -1) (0, -1) (1, -1)
    // (-1,  0) (0,  0) (1,  0)
    // (-1,  1) (0,  1) (1,  1)
    const uint offset = (filter_size - 1) / 2;
    const uint filter_length = filter_size * filter_size;

    #ifdef LENA
        for (uint y = 0; y < height; ++y) {
            for (uint x = 0; x < width; ++x) {
                // Print out Lena
                cout << static_cast<int>(input[x + width * y]) << " ";
            }
            cout << endl;
        }
    #endif

    /**
     * Iterate and perform median filter analysis on every pixel.
     * Make outer loop the ys so that successive reads are as close to each other as possible,
     * i.e. for single-threaded CPU code, it is most important for caching, but for GPUs it is
     * most important for coalesced memory access (and maybe caching).
     * If we iterate over the rows first, we have 0 coalescing then.
     */
    for (uint y = 0; y < height; ++y) {
        for (uint x = 0; x < width; ++x) {

            uchar filter_array[filter_length];
            // Init the filter array with 0 or 255 values
            // Will write over the indices that are VIEWABLE from the context pixel
            for (uint i = 0; i < filter_length; ++i) {
                filter_array[i] = i % 2 == 0 ? MIN_RGB_VALUE : MAX_RGB_VALUE;
            }

            // What pixel am I currently looking at
            const uchar * context  = &input[x + width * y];
            uchar * output_context = &output[x + width * y];

    	    // Populate the filter_array.
            uint filter_array_index = 0;

            for (int y_offset = -1 * static_cast<int>(offset); y_offset <= static_cast<int>(offset); ++y_offset) {
                for (int x_offset = -1 * static_cast<int>(offset); x_offset <= static_cast<int>(offset); ++x_offset) {
        		    // Handle special cases for when the offset would place us beyond the bounds of the input.
                    const int x_focus = x + x_offset;
                    const int y_focus = y + y_offset;

                    // Check if one of the neighboring pixels of our context pixel is outside the grid
                    if (x_focus < 0 || x_focus >= width || y_focus < 0 || y_focus >= height) {
                        continue;
                    }
                    // Otherwise we're not an edge or corner, so we have all of our neighbors
                    filter_array[filter_array_index++] = *(context + static_cast<int>(x_offset) + static_cast<int>(width) * static_cast<int>(y_offset));
        		}
    	    }

    	    // Sort the filter_array.
            sort(filter_array, filter_array + filter_length);

            // Print the filter array to test.
            for (uint i = 0; i < filter_length - 1; ++i) {
                cout << static_cast<int>(filter_array[i]) << " ";
            }
            cout << static_cast<int>(filter_array[filter_length - 1]) << endl;

            // Grab the median. Note that the since we always had odd window sizes,
            // then filter_size * filter_size is always odd as well - so no need to
            // handle special cases for even or odd number for the median.
            *output_context = filter_array[(filter_length - 1) / 2];
	   }
    }
}
